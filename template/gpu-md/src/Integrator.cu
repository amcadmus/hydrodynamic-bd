#include "hip/hip_runtime.h"
#include "MDSystem_interface.h"
#include "common.h"
#include "Integrator.h"
#include "Auxiliary.h"
#include <stdio.h>
#include "Statistic_interface.h"


__global__ void leapFrog1Step (const IndexType numAtom,
			       const ScalorType * massi,
			       CoordType * coord,
			       ScalorType * velox,
			       ScalorType * veloy, 
			       ScalorType * veloz,
			       const ScalorType * forcx,
			       const ScalorType * forcy, 
			       const ScalorType * forcz,
			       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii >= numAtom) return;
  ScalorType mi = massi[ii];
  ScalorType vx, vy, vz;
  vx = (velox[ii] += dt * forcx[ii] * mi);
  coord[ii].x += dt * vx;
  vy = (veloy[ii] += dt * forcy[ii] * mi);
  coord[ii].y += dt * vy;
  vz = (veloz[ii] += dt * forcz[ii] * mi);
  coord[ii].z += dt * vz;
}

__global__ void leapFrog1Step (const IndexType numAtom,
			       const ScalorType * mass,
			       const ScalorType * massi,
			       CoordType * coord,
			       ScalorType * velox,
			       ScalorType * veloy, 
			       ScalorType * veloz,
			       const ScalorType * forcx,
			       const ScalorType * forcy, 
			       const ScalorType * forcz,
			       const ScalorType dt,
			       ScalorType * statistic_buffxx,
			       ScalorType * statistic_buffyy,
			       ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  ScalorType vx, vy, vz;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    vx = (velox[ii] += dt * forcx[ii] * mi);
    coord[ii].x += dt * vx;
    vy = (veloy[ii] += dt * forcy[ii] * mi);
    coord[ii].y += dt * vy;
    vz = (veloz[ii] += dt * forcz[ii] * mi);
    coord[ii].z += dt * vz;
  }

  extern __shared__ volatile ScalorType buff [];

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}

#ifndef COORD_IN_ONE_VEC
__global__ void leapFrogStepX (const IndexType numAtom,
			       const ScalorType * massi,
			       ScalorType * coordx,
			       ScalorType * coordy, 
			       ScalorType * coordz,
			       const ScalorType * velox,
			       const ScalorType * veloy, 
			       const ScalorType * veloz,
			       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii >= numAtom) return;
  coordx[ii] += dt * velox[ii];
  coordy[ii] += dt * veloy[ii];
  coordz[ii] += dt * veloz[ii];
}
#else
__global__ void leapFrogStepX (const IndexType numAtom,
			       const ScalorType * massi,
			       CoordType * coord,
			       const ScalorType * velox,
			       const ScalorType * veloy, 
			       const ScalorType * veloz,
			       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii >= numAtom) return;
  coord[ii].x += dt * velox[ii];
  coord[ii].y += dt * veloy[ii];
  coord[ii].z += dt * veloz[ii];
}
#endif


__global__ void leapFrogStepV (const IndexType numAtom,
			       const ScalorType * massi,
			       ScalorType * velox,
			       ScalorType * veloy, 
			       ScalorType * veloz,
			       const ScalorType * forcx,
			       const ScalorType * forcy, 
			       const ScalorType * forcz,
			       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    (velox[ii] += dt * forcx[ii] * mi);
    (veloy[ii] += dt * forcy[ii] * mi);
    (veloz[ii] += dt * forcz[ii] * mi);
  }
}

__global__ void leapFrogStepV (const IndexType numAtom,
			       const ScalorType * mass,
			       const ScalorType * massi,
			       ScalorType * velox,
			       ScalorType * veloy, 
			       ScalorType * veloz,
			       const ScalorType * forcx,
			       const ScalorType * forcy, 
			       const ScalorType * forcz,
			       const ScalorType dt,
			       ScalorType * statistic_buffxx,
			       ScalorType * statistic_buffyy,
			       ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  ScalorType vx, vy, vz;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    vx = (velox[ii] += dt * forcx[ii] * mi);
    vy = (veloy[ii] += dt * forcy[ii] * mi);
    vz = (veloz[ii] += dt * forcz[ii] * mi);
  }

  extern __shared__ volatile ScalorType buff [];

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}


__device__ IndexType integrator_counter_prepare_x = 0;
__device__ IndexType integrator_counter_prepare_y = 0;
__device__ IndexType integrator_counter_prepare_z = 0;
__global__ void initRemoveTranslationalFreedom ()
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  if (tid + bid == 0){
    integrator_counter_prepare_x = 0;
    integrator_counter_prepare_y = 0;
    integrator_counter_prepare_z = 0;
  }
}

__global__ void prepareRemoveTranslationalFreedom (IndexType numAtom,
						   ScalorType * mass,
						   ScalorType * velox,
						   ScalorType * veloy,
						   ScalorType * veloz,
						   ScalorType * buffx,
						   ScalorType * buffy,
						   ScalorType * buffz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = threadIdx.x + bid * blockDim.x;

  extern __shared__ volatile ScalorType buff[];
  buff[tid] = 0.f;
  buff[tid+blockDim.x] = 0.f;
  __syncthreads();
  if (ii < numAtom){
    buff[tid] = mass[ii] * velox[ii];
  }
  __syncthreads();
  sumVectorBlockBuffer (buff, blockDim.x);
  if (tid == 0) buffx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[tid] = mass[ii] * veloy[ii];
  }
  __syncthreads();
  sumVectorBlockBuffer (buff, blockDim.x);
  if (tid == 0) buffy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[tid] = mass[ii] * veloz[ii];
  }
  __syncthreads();
  sumVectorBlockBuffer (buff, blockDim.x);
  if (tid == 0) buffz[bid] = buff[0];
}

__global__ void removeFreedom (IndexType numAtom,
			       ScalorType * velox, 
			       ScalorType * veloy,
			       ScalorType * veloz,
			       ScalorType totalMassi,
			       ScalorType * sums)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  if (ii < numAtom) {
    velox[ii] -= sums[0] * totalMassi;
    veloy[ii] -= sums[1] * totalMassi;
    veloz[ii] -= sums[2] * totalMassi;
  }
}


#ifndef COORD_IN_ONE_VEC
__global__ void velocityVerlet_part1 (const IndexType numAtom,
				      const ScalorType * massi,
				      TypeType * type,
				      TypeType freezType,
				      ScalorType * coordx,
				      ScalorType * coordy, 
				      ScalorType * coordz,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;
  
  ScalorType hdtmi = 0.5f*dt*massi[ii];
  velox[ii]  += hdtmi * forcx[ii];
  veloy[ii]  += hdtmi * forcy[ii];
  veloz[ii]  += hdtmi * forcz[ii];
  if (type[ii] != freezType) {
    coordx[ii] += dt * velox[ii];
    coordy[ii] += dt * veloy[ii];
    coordz[ii] += dt * veloz[ii];
  }
  else {
    velox[ii] = 0.;
    veloy[ii] = 0.;
    veloz[ii] = 0.;
  }
}
#else
__global__ void velocityVerlet_part1 (const IndexType numAtom,
				      const ScalorType * massi,
				      TypeType * type,
				      TypeType freezType,
				      CoordType * coord,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;
  
  ScalorType hdtmi = 0.5f*dt*massi[ii];
  velox[ii]  += hdtmi * forcx[ii];
  veloy[ii]  += hdtmi * forcy[ii];
  veloz[ii]  += hdtmi * forcz[ii];
  if (type[ii] != freezType) {
    coord[ii].x += dt * velox[ii];
    coord[ii].y += dt * veloy[ii];
    coord[ii].z += dt * veloz[ii];
  }
  else {
    velox[ii] = 0.;
    veloy[ii] = 0.;
    veloz[ii] = 0.;
  }
}
#endif

#ifndef COORD_IN_ONE_VEC
__global__ void velocityVerlet_part1 (const IndexType numAtom,
				      const ScalorType * massi,
				      ScalorType * coordx,
				      ScalorType * coordy, 
				      ScalorType * coordz,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;
  
  ScalorType hdtmi = 0.5f*dt*massi[ii];
  velox[ii]  += hdtmi * forcx[ii];
  veloy[ii]  += hdtmi * forcy[ii];
  veloz[ii]  += hdtmi * forcz[ii];
  coordx[ii] += dt * velox[ii];
  coordy[ii] += dt * veloy[ii];
  coordz[ii] += dt * veloz[ii];
}
#else
__global__ void velocityVerlet_part1 (const IndexType numAtom,
				      const ScalorType * massi,
				      CoordType * coord,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;
  
  ScalorType hdtmi = 0.5f*dt*massi[ii];
  velox[ii]  += hdtmi * forcx[ii];
  veloy[ii]  += hdtmi * forcy[ii];
  veloz[ii]  += hdtmi * forcz[ii];
  coord[ii].x += dt * velox[ii];
  coord[ii].y += dt * veloy[ii];
  coord[ii].z += dt * veloz[ii];
}
#endif


__global__ void velocityVerlet_part2 (const IndexType numAtom,
				      const ScalorType * massi,
				      TypeType * type,
				      TypeType freezType,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;
  
  ScalorType hdtmi = 0.5f*dt*massi[ii];
  if (type[ii] != freezType) {
    velox[ii] += hdtmi * forcx[ii];
    veloy[ii] += hdtmi * forcy[ii];
    veloz[ii] += hdtmi * forcz[ii];
  }
  else {
    velox[ii] = 0.;
    veloy[ii] = 0.;
    veloz[ii] = 0.;
  }
}


__global__ void velocityVerlet_part2 (const IndexType numAtom,
				      const ScalorType * mass,
				      const ScalorType * massi,
				      TypeType * type,
				      TypeType freezType,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt,
				      ScalorType * statistic_buffxx,
				      ScalorType * statistic_buffyy,
				      ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  extern __shared__ volatile ScalorType buff [];
ScalorType vx(0.f), vy(0.f), vz(0.f);
  if (ii < numAtom) {
    if (type[ii] != freezType) {
      ScalorType hdtmi = 0.5f*dt*massi[ii];
      vx = (velox[ii] += hdtmi * forcx[ii]);
      vy = (veloy[ii] += hdtmi * forcy[ii]);
      vz = (veloz[ii] += hdtmi * forcz[ii]);
    }
    else {
      vx = (velox[ii] = 0.);
      vy = (veloy[ii] = 0.);
      vz = (veloz[ii] = 0.);
    }
  }

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}


__global__ void velocityVerlet_part2 (const IndexType numAtom,
				      const ScalorType * massi,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;

  ScalorType hdtmi = 0.5f*dt*massi[ii];
  velox[ii] += hdtmi * forcx[ii];
  veloy[ii] += hdtmi * forcy[ii];
  veloz[ii] += hdtmi * forcz[ii];
}


__global__ void velocityVerlet_part2 (const IndexType numAtom,
				      const ScalorType * mass,
				      const ScalorType * massi,
				      ScalorType * velox,
				      ScalorType * veloy, 
				      ScalorType * veloz,
				      const ScalorType * forcx,
				      const ScalorType * forcy, 
				      const ScalorType * forcz,
				      const ScalorType dt,
				      ScalorType * statistic_buffxx,
				      ScalorType * statistic_buffyy,
				      ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  extern __shared__ volatile ScalorType buff [];
ScalorType vx(0.f), vy(0.f), vz(0.f);
  if (ii < numAtom) {
    ScalorType hdtmi = 0.5f*dt*massi[ii];
    vx = (velox[ii] += hdtmi * forcx[ii]);
    vy = (veloy[ii] += hdtmi * forcy[ii]);
    vz = (veloz[ii] += hdtmi * forcz[ii]);
  }

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}


__global__ void velocityVerlet_part2a (const IndexType numAtom,
				       const ScalorType * mass,
				       const ScalorType * massi,
				       ScalorType * velox,
				       ScalorType * veloy, 
				       ScalorType * veloz,
				       const ScalorType * forcx,
				       const ScalorType * forcy, 
				       const ScalorType * forcz,
				       const ScalorType dt,
				       ScalorType * statistic_buff)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  ScalorType vx, vy, vz;
  if (ii < numAtom) { 
    ScalorType hdtmi = 0.5f*dt*massi[ii];
    vx = (velox[ii] += hdtmi * forcx[ii]);
    vy = (veloy[ii] += hdtmi * forcy[ii]);
    vz = (veloz[ii] += hdtmi * forcz[ii]);
  }

  extern __shared__ volatile ScalorType buff [];
  if (ii < numAtom)
    buff[tid] = 0.5 * mass[ii] * (vx*vx + vy*vy + vz*vz);
  else
    buff[tid] = 0.f;
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buff[bid] = buff[0];  
}


__global__ void
Langevin_BAOAB_partAOA (const ScalorType pref0,
			const ScalorType pref1,
			const TypeType t0,
			const TypeType t1,
			const IndexType numAtom,
			const ScalorType * massi,
			const TypeType * type,
			CoordType * coord,
			ScalorType * velox,
			ScalorType * veloy, 
			ScalorType * veloz,
			const ScalorType * forcx,
			const ScalorType * forcy, 
			const ScalorType * forcz,
			const ScalorType * random_x,
			const ScalorType * random_y,
			const ScalorType * random_z,
			const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;

  if (ii >= numAtom) return;
  
  ScalorType hdt = 0.5f*dt;
  ScalorType sqrtmi = sqrtf(massi[ii]);
  TypeType mt = type[ii];

  coord[ii].x += hdt * velox[ii];
  coord[ii].y += hdt * veloy[ii];
  coord[ii].z += hdt * veloz[ii];

  if (mt == t0 || mt == t1){
    velox[ii] = pref0 * velox[ii] + pref1 * sqrtmi * random_x[ii];
    veloy[ii] = pref0 * veloy[ii] + pref1 * sqrtmi * random_y[ii];
    veloz[ii] = pref0 * veloz[ii] + pref1 * sqrtmi * random_z[ii];
  }
  
  coord[ii].x += hdt * velox[ii];
  coord[ii].y += hdt * veloy[ii];
  coord[ii].z += hdt * veloz[ii];
}


__global__ void
Langevin_BAOAB_partB (const IndexType numAtom,
		      const ScalorType * massi,
		      ScalorType * velox,
		      ScalorType * veloy, 
		      ScalorType * veloz,
		      const ScalorType * forcx,
		      const ScalorType * forcy, 
		      const ScalorType * forcz,
		      const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  if (ii >= numAtom) return;

  ScalorType hdt = 0.5f*dt;
  ScalorType mi = massi[ii];
  
  velox[ii] += hdt * mi * forcx[ii];
  veloy[ii] += hdt * mi * forcy[ii];
  veloz[ii] += hdt * mi * forcz[ii];
}

__global__ void
Langevin_BAOAB_partB (const IndexType numAtom,
		      const ScalorType * massi,
		      ScalorType * velox,
		      ScalorType * veloy, 
		      ScalorType * veloz,
		      const ScalorType * forcx,
		      const ScalorType * forcy, 
		      const ScalorType * forcz,
		      const ScalorType dt,
		      ScalorType * statistic_buffxx,
		      ScalorType * statistic_buffyy,
		      ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;

  extern __shared__ volatile ScalorType buff [];
  ScalorType vx(0.f), vy(0.f), vz(0.f);

  if (ii < numAtom) {
    ScalorType hdt = 0.5f*dt;
    ScalorType mi = massi[ii];

    velox[ii] += hdt * mi * forcx[ii];
    veloy[ii] += hdt * mi * forcy[ii];
    veloz[ii] += hdt * mi * forcz[ii];

    vx = velox[ii];
    vy = veloy[ii];
    vz = veloz[ii];
  }

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f / massi[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}


__global__ void velocityRescale_rescale (const IndexType numAtom,
					 ScalorType * velox,
					 ScalorType * veloy, 
					 ScalorType * veloz,
					 const ScalorType alpha)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  
  if (ii < numAtom) {
    (velox[ii] *= alpha);
    (veloy[ii] *= alpha);
    (veloz[ii] *= alpha);
  }
}

__global__ void velocityRescale_rescale (const IndexType numAtom,
					 const ScalorType * mass,
					 ScalorType * velox,
					 ScalorType * veloy, 
					 ScalorType * veloz,
					 const ScalorType alpha,
					 ScalorType * statistic_buffxx,
					 ScalorType * statistic_buffyy,
					 ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType tid = threadIdx.x;
  IndexType ii = tid + bid * blockDim.x;
  ScalorType vx, vy, vz;
  
  if (ii < numAtom) {
    vx = (velox[ii] *= alpha);
    vy = (veloy[ii] *= alpha);
    vz = (veloz[ii] *= alpha);
  }

  extern __shared__ volatile ScalorType buff [];

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5 * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}



__global__ void rescaleData (const IndexType numAtom,
			     ScalorType * data,
			     ScalorType alpha)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  
  if (ii < numAtom) data[ii] *= alpha;
}



__global__ void
leapFrogStepV_VCouple (const IndexType numAtom,
		       const ScalorType * massi,
		       ScalorType * velox,
		       ScalorType * veloy, 
		       ScalorType * veloz,
		       const ScalorType * forcx,
		       const ScalorType * forcy, 
		       const ScalorType * forcz,
		       const ScalorType lambda0,
		       const ScalorType lambda1,
		       const ScalorType lambda2,
		       const ScalorType dt)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    ScalorType hdt = 0.5f * dt;
    ScalorType tmp = 1.f - hdt * lambda0;
    ScalorType tmp1= 1.f + hdt * lambda0;
    velox[ii] = (tmp * velox[ii] + dt * mi * forcx[ii]) / tmp1;
    tmp = 1.f - hdt * lambda1;
    tmp1= 1.f + hdt * lambda1;
    veloy[ii] = (tmp * veloy[ii] + dt * mi * forcy[ii]) / tmp1;
    tmp = 1.f - hdt * lambda2;
    tmp1= 1.f + hdt * lambda2;
    veloz[ii] = (tmp * veloz[ii] + dt * mi * forcz[ii]) / tmp1;
  }
}

__global__ void
leapFrogStepV_VCouple (const IndexType numAtom,
		       const ScalorType * mass,
		       const ScalorType * massi,
		       ScalorType * velox,
		       ScalorType * veloy, 
		       ScalorType * veloz,
		       const ScalorType * forcx,
		       const ScalorType * forcy, 
		       const ScalorType * forcz,
		       const ScalorType lambda0,
		       const ScalorType lambda1,
		       const ScalorType lambda2,
		       const ScalorType dt,
		       ScalorType * statistic_buffxx,
		       ScalorType * statistic_buffyy,
		       ScalorType * statistic_buffzz)
{
  IndexType bid = blockIdx.x + gridDim.x * blockIdx.y;
  IndexType ii = threadIdx.x + bid * blockDim.x;
  ScalorType vx, vy, vz;
  
  if (ii < numAtom) {
    ScalorType mi = massi[ii];
    ScalorType hdt = 0.5f * dt;
    ScalorType tmp = 1.f - hdt * lambda0;
    ScalorType tmp1= 1.f + hdt * lambda0 ;
    vx = (velox[ii] = ((tmp * velox[ii] + dt * mi * forcx[ii]) / tmp1));
    tmp = 1.f - hdt * lambda1;
    tmp1= 1.f + hdt * lambda1;
    vy = (veloy[ii] = ((tmp * veloy[ii] + dt * mi * forcy[ii]) / tmp1));
    tmp = 1.f - hdt * lambda2;
    tmp1= 1.f + hdt * lambda2;
    vz = (veloz[ii] = ((tmp * veloz[ii] + dt * mi * forcz[ii]) / tmp1));
  }

  extern __shared__ volatile ScalorType buff [];

  ScalorType scalor;
  if (ii < numAtom) scalor = 0.5f * mass[ii];
  else scalor = 0.f;
  
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vx * vx;
  }
  else {
    buff[threadIdx.x] = 0.f;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffxx[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vy * vy;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffyy[bid] = buff[0];
  __syncthreads();
  if (ii < numAtom){
    buff[threadIdx.x] = scalor * vz * vz;
  }
  sumVectorBlockBuffer_2 (buff);
  if (threadIdx.x == 0) statistic_buffzz[bid] = buff[0];
}


