#include "hip/hip_runtime.h"
#include <stdio.h>
#include "MDSystem_interface.h"
#include "common.h"
#include "BoxGeometry.h"
#include "MDSystem.h"
#include "RandomGenerator.h"
#include "Auxiliary.h"
#include "NeighborList_interface.h"
#include"Statistic.h"
#include "Integrator_interface.h"
#include "InteractionEngine_interface.h"
#include "Reshuffle_interface.h"
#include "Displacement_interface.h"

#include "Topology.h"
#include "SystemBondedInteraction.h"

#include "BondInteraction.h"
#include "NonBondedInteraction.h"


// #define NThreadsPerBlockCell	32
// #define NThreadsPerBlockAtom	4

#define NThreadsPerBlockCell	160
#define NThreadsPerBlockAtom	96

void posiRenormalize (int * posi,
		      const int size)
{
  while ((*posi) < 0) {
    *posi += size;
  }
  while ((*posi) >= size) {
    *posi -= size;
  }
}

double calDeltaF (const MDSystem & sys,
		  const double & kk,
		  const char * liquidName,
		  double * corrDeltaF,
		  const int corrSize,
		  int * corrDeltaFPosi,
		  int * corrDeltaFNvalid)
{
  double sumFup = 0; 
  double sumFdown = 0;
  double boxc = sys.box.size.z * 0.5;
  
  for (int ii = 0; ii < sys.hdata.numAtom; ++ii){
    if (strcmp(&(sys.hdata.atomName[ii*StringSize]), liquidName) == 0) {
      if ( sys.hdata.coord[ii].z > sys.box.size.z * 0.5 ){
	sumFup += sys.hdata.recordForcx[ii] * sin( kk * (sys.hdata.coord[ii].z - boxc) );
      }
      else {
	sumFdown += sys.hdata.recordForcx[ii] * sin( kk * (sys.hdata.coord[ii].z - boxc) );
      }
    }
  }

//  printf ("%f   ", sumFup - sumFdown);
  corrDeltaF[*corrDeltaFPosi] = (sumFup - sumFdown) * 0.5;
  (*corrDeltaFPosi) ++;
  posiRenormalize (corrDeltaFPosi, corrSize);
  
  if ((*corrDeltaFNvalid) < corrSize) {
    (*corrDeltaFNvalid) ++;
  }

  return (sumFup - sumFdown) * 0.5;
}

void depositData (const double * corrDeltaF,
		  const int corrSize,
		  const int corrDeltaFPosi,
		  const int corrDeltaFNvalid,
		  double * corrSumData,
		  int * corrSumDataCount,
		  double * corrData,
		  int * corrDataCount)
{
  int start = corrDeltaFPosi - 1;
  int end = corrDeltaFPosi - corrDeltaFNvalid - 1; 
  posiRenormalize (&start, corrSize);
  posiRenormalize (&end, corrSize);
  
  int ii = start;
  corrSumData[0] += corrDeltaF[start];
//  printf (" %f\n", corrDeltaF[start]);
  corrSumDataCount[0] ++;
  corrData[0] += corrDeltaF[start] * corrDeltaF[start];
  corrDataCount[0] ++;
  ii --;
  posiRenormalize (&ii, corrSize);
  int count = 1;
  
  while (ii != end){
    corrData[count] += corrDeltaF[ii] * corrDeltaF[start];
    corrDataCount[count] ++;
    --ii;
    ++count;
    posiRenormalize(&ii, corrSize);
  }
}


int main(int argc, char * argv[])
{
  IndexType nstep = 100000;
  IndexType confFeq = 250;
  IndexType thermoFeq = 5000;
  ScalorType rcut = 2.5;
  ScalorType nlistExten = 0.5;
  ScalorType refT = 2.80;
  ScalorType tauT = 0.1;
  char * filename;
  InteractionType recordType = 1;
  IndexType numType0 = 0;
  IndexType numType1 = 0;
  IndexType numType2 = 0;
  IndexType num_w = 0;
  IndexType num_f = 0;
  double kk = 0.224214;
  ScalorType lattice_k = 900.;
  
  if (argc != 4){
    printf ("Usage:\n%s conf.gro nstep device\n", argv[0]);
    return 1;
  }
  if (argc != 1){
    nstep = atoi(argv[2]);
    filename = argv[1];
  }
  printf ("# setting device to %d\n", atoi(argv[3]));
  hipSetDevice (atoi(argv[3]));
  checkCUDAError ("set device");

  IndexType corrFeq = 1;
  IndexType corrStart = nstep / 10;
  IndexType corrNstep = 10000 + 1;
  double * corrDeltaF = (double *) malloc (sizeof(double) * corrNstep);
  double * corrData = (double *) malloc (sizeof(double) * corrNstep);
  int * corrDataCount = (int *) malloc (sizeof(int) * corrNstep);
  double corrSumData = 0.;
  int corrSumDataCount = 0;
  int corrDeltaFPosi = 0;
  int corrDeltaFNvalid = 0;
  for (int ii = 0; ii < corrNstep; ++ii){
    corrDeltaF[ii] = 0.;
    corrData[ii] = 0.;
    corrDataCount[ii] = 0;
  }
  double sumDF = 0.;
  double sumD2F = 0.;
  int sumD2FCount = 0;

  MDSystem sys;
  sys.initConfig(filename);
  // double com = 0;
  // for (int ii = 0; ii < sys.hdata.numAtom; ++ii){
  //   com += sys.hdata.coord[ii].z;
  // }
  // com /= double (sys.hdata.numAtom);
  // com = sys.box.size.z * 0.5 - com;
  // for (int ii = 0; ii < sys.hdata.numAtom; ++ii){
  //   sys.hdata.coord[ii].z += com;
  // }  
  
  unsigned position = 0;
  for (unsigned ii = position; ii < sys.hdata.numAtom ; ++ii){
    if (strcmp(&(sys.hdata.atomName[ii*StringSize]), "tp01") == 0) {
      continue;
    }
    else {
      numType0 = ii;
      break;
    }
  }
  position = numType0;
  for (unsigned ii = position; ii < sys.hdata.numAtom ; ++ii){
    if (strcmp(&(sys.hdata.atomName[ii*StringSize]), "tp02") == 0) {
      continue;
    }
    else {
      numType1 = ii;
      break;
    }
  }
  numType1 -= position;
  numType2 = sys.hdata.numAtom - numType1 - numType0;
  num_w = numType0 + numType1;
  num_f = numType2;
  printf ("# num tp01: %d tp02: %d ty03: %d   num wall: %d num fluid: %d\n",
	  numType0, numType1, numType2,
	  num_w, num_f);

  double minz = 1e10;
  for (int ii = 0; ii < numType0; ++ii){
    if (sys.hdata.coord[ii].z < minz){
      minz = sys.hdata.coord[ii].z;
    }
  }
  double maxz = -1;
  for (int ii = numType0; ii < numType0 + numType1; ++ii) {
    if (sys.hdata.coord[ii].z > maxz){
      maxz = sys.hdata.coord[ii].z;
    }
  }
  double shift = 0.5 * sys.box.size.z - 0.5 * (maxz + minz);
  // for (int ii = 0; ii < sys.hdata.numAtom; ++ii){
  //   sys.hdata.coord[ii].z += shift;
  // }
  printf ("# minz is %f   maxz is %f   shift the system by %f\n", minz, maxz, shift);

  TypeType freezType ;
  Topology::System sysTop;
  Topology::Molecule mol_w;
  mol_w.pushAtom (Topology::Atom (1.0, 0.0, 0));
  freezType = 100;
  Topology::Molecule mol_f;
  mol_f.pushAtom (Topology::Atom (1.0, 0.0, 1));

  sysTop.addMolecules (mol_w, num_w);
  sysTop.addMolecules (mol_f, num_f);
  
  LennardJones6_12Parameter ljparam_ww;
  ljparam_ww.reinit (1.00f, 1.00f, 1.00f, 0.f, rcut);
  LennardJones6_12Parameter ljparam_ff;
  ljparam_ff.reinit (1.00f, 1.00f, 1.00f, 0.f, rcut);
  LennardJones6_12Parameter ljparam_fw;
  ljparam_fw.reinit (1.16f, 1.04f, 0.70f, 0.f, rcut);

//  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(0, 0, ljparam_ww));
  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(0, 1, ljparam_fw));
  sysTop.addNonBondedInteraction (Topology::NonBondedInteraction(1, 1, ljparam_ff));

  sys.initTopology (sysTop);
  sys.initDeviceData ();
  
  SystemNonBondedInteraction sysNbInter;
  sysNbInter.reinit (sysTop);
  ScalorType energyCorr = sysNbInter.energyCorrection ();
  ScalorType pressureCorr = sysNbInter.pressureCorrection ();
  sysNbInter.printTable ();
  
  ScalorType maxrcut = sysNbInter.maxRcut();
  ScalorType rlist = maxrcut + nlistExten;
  CellList clist (sys, rlist, NThreadsPerBlockCell, NThreadsPerBlockAtom);
  NeighborList nlist (sysNbInter, sys, rlist, NThreadsPerBlockAtom, 2.f);
  sys.normalizeDeviceData ();
  clist.rebuild (sys, NULL);
  nlist.rebuild (sys, clist, NULL);
  Displacement_max disp (sys, NThreadsPerBlockAtom);
  disp.recordCoord (sys);
  
  MDStatistic st(sys);
  TranslationalFreedomRemover tfremover (sys, NThreadsPerBlockAtom);
  InteractionEngine inter (sys, NThreadsPerBlockAtom);
  inter.registNonBondedInteraction (sysNbInter);
  // inter.applyNonBondedInteraction (sys, nlist, recordType, st, NULL, &timer);
  // inter.applyLatticeInteraction (sys, lattice_k, 0, NULL);
  
  MDTimer timer;
  unsigned i;
  ScalorType dt = 0.002;
  ScalorType seed = 1;
  RandomGenerator_MT19937::init_genrand (seed);

  VelocityVerlet inte_vv (sys, NThreadsPerBlockAtom);
  VelocityRescale inte_vr (sys, NThreadsPerBlockAtom, refT, 0.1);
  NoseHoover_Chains2 nhc;
  // nhc.reinit (sys, num_w, NThreadsPerBlockAtom, refT, tauT);
  nhc.reinit (sys, 0, NThreadsPerBlockAtom, refT, tauT);

  Reshuffle resh (sys);
  
  timer.tic(mdTimeTotal);
  if (resh.calIndexTable (clist, &timer)){
    sys.reshuffle   (resh.indexTable, sys.hdata.numAtom, &timer);
    clist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    nlist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
    disp.reshuffle  (resh.indexTable, sys.hdata.numAtom, &timer);  
  }
  
  printf ("# prepare ok, start to run\n");
  sys.recoverDeviceData (&timer);
  sys.updateHostFromRecovered (&timer);
  sys.writeHostDataGro ("confstart.gro", 0, 0.f, &timer);
  printf ("# prepare ok, start to run\n");
  printf ("#*     1     2           3         4            5       6                7        8   9\n");
  printf ("#* nstep  time  nonBondedE  kineticE  temperature  totalE  NHC_Hamiltonian pressure box\n");

  try{
    sys.initWriteXtc ("traj.xtc");
    sys.initWriteTrr ("traj.trr");
    sys.recoverDeviceData (&timer);
    sys.updateHostFromRecovered (&timer);
    sys.writeHostDataXtc (0, 0*dt, &timer);
    sys.writeHostDataTrr (0, 0*dt, &timer);
    for (i = 0; i < nstep; ++i){
      if (i%10 == 0){
//	tfremover.remove (sys, &timer);
      }
      
      nhc.operator_L (0.5 * dt, sys, &timer);
      inte_vv.step1 (sys, freezType, dt, &timer);

      st.clearDevice();
      ScalorType maxdr = disp.calMaxDisplacemant (sys, &timer);
      if (maxdr > nlistExten * 0.5){
	// printf ("# Rebuild at step %09i ... ", i+1);
	// fflush(stdout);
	// rebuild
	sys.normalizeDeviceData (&timer);
	disp.recordCoord (sys);
	clist.rebuild (sys, &timer);
	nlist.rebuild (sys, clist, &timer);
	// printf ("done\n");
	// fflush(stdout);
      }
      inter.clearInteraction (sys);
      inter.applyNonBondedInteraction (sys, nlist, recordType, st, NULL, &timer);
      inter.applyLatticeInteraction (sys, lattice_k, 0, NULL);

      inte_vv.step2 (sys, freezType, dt, &timer);
      if ((i+1) % thermoFeq == 0){	
	nhc.operator_L (0.5 * dt, sys, st, &timer);
      }
      else {
	nhc.operator_L (0.5 * dt, sys, &timer);	
      }      

      if ((i+1) % thermoFeq == 0){
	st.updateHost ();
	printf ("%09d %07e %.7e %.7e %.7e %.7e %.7e %.7e %.7e %.2e %.2e %.2e %.2e\n",
		(i+1),  
		(i+1) * dt, 
		st.nonBondedEnergy(),
		st.kineticEnergy(),
		st.kineticEnergy() * 2. / 3. / (double (num_f) - 3.),
		st.nonBondedEnergy() +
		st.kineticEnergy(),
		st.nonBondedEnergy() +
		st.kineticEnergy() +
		nhc.HamiltonianContribution (),
		st.pressure(sys.box),
		sys.box.size.x,
		nhc.xi1,
		nhc.vxi1,
		nhc.xi2,
		nhc.vxi1
	    );
	fflush(stdout);
      }

      if (i+1 >= corrStart && (i+1) % confFeq == 0){
      	sys.recoverDeviceData (&timer);
      	sys.updateHostFromRecovered (&timer);
      	sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
      	sys.writeHostDataTrr (i+1, (i+1)*dt, &timer);
      }
      // if (i+1 >= corrStart && (i+1) % corrFeq == 0){
      // 	sys.recoverDeviceData (&timer);
      // 	sys.updateHostFromRecovered (&timer);
      // 	double DF = calDeltaF (sys, kk, "tp03",
      // 			       corrDeltaF, corrNstep,
      // 			       &corrDeltaFPosi, &corrDeltaFNvalid);
      // 	sumDF += DF;
      // 	sumD2F += DF * DF;
      // 	sumD2FCount++;
      // 	depositData (corrDeltaF, corrNstep, corrDeltaFPosi, corrDeltaFNvalid,
      // 		     &corrSumData, &corrSumDataCount,
      // 		     corrData, corrDataCount);
      // }

      if ((i+1) % 100 == 0){
	if (resh.calIndexTable (clist, &timer)){
	  sys.reshuffle   (resh.indexTable, sys.hdata.numAtom, &timer);
	  clist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
	  nlist.reshuffle (resh.indexTable, sys.hdata.numAtom, &timer);  
	  disp.reshuffle  (resh.indexTable, sys.hdata.numAtom, &timer);  
	}
      }
    }
    sys.endWriteXtc();
    sys.endWriteTrr();
    sys.recoverDeviceData (&timer);
    sys.updateHostFromRecovered (&timer);
    sys.writeHostDataGro ("confout.gro", nstep, nstep*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
  }
  catch (MDExcptCuda & e){
    // resh.recoverMDDataToHost (sys, &timer);
    // sys.writeHostDataXtc (i+1, (i+1)*dt, &timer);
    timer.toc(mdTimeTotal);
    timer.printRecord (stderr);
    return 1;
  }
  catch (MDException &e){
    fprintf (stderr, "%s\n", e.what());
    return 1;
  }


  FILE * fp;
  if ((fp = fopen("corr.out","w")) == NULL) {
    printf ("cannot open file %s\n", "corr.out");
  }
  for (int ii = 0; ii < corrNstep; ++ii){
    if (corrDataCount[ii] != 0){
      double tmpa = corrData[ii] / double(corrDataCount[ii]);
      double tmpb = corrSumData  / double(corrSumDataCount);
      fprintf (fp, "%f %e   %e %e\n", ii * dt * corrFeq, tmpa - tmpb * tmpb, tmpa, tmpb);
    }
    else {
      fprintf (fp, "%f %e\n", ii * dt * corrFeq, 0.);
    }
  }
  free (corrDeltaF);
  free (corrData);
  free (corrDataCount);

  printf ("avg sumD2F: %f, avg sumDF: %f\n",
	  sumD2F / double(sumD2FCount),
	  sumDF / double (sumD2FCount)
      );
  
  return 0;
}

  
